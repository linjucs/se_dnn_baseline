#include "hip/hip_runtime.h"
#include "DevFunc.h"
#include <stdlib.h>

__global__ void kernBinary(int n, float* in_vec, float* rand_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < n)
    {
			if(in_vec[i] > rand_vec[i])
			{
				in_vec[i] = 1.0f;
			}
			else
			{
				in_vec[i] = 0.0f;
			}
		}
}

__global__ void kernWeightMultiP( int n, float p, float* in_vec )
{
//	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
//	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
//	if(i < prev_n&& j < cur_n)
//	{
//	   in_vec[i+cur_n*j] = in_vec[i+cur_n*j]*p;
//	}
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if( i < n )
	{
		in_vec[i]=in_vec[i]*p;
	}
}
__global__ void kernDropout(int n, float p ,float* in, float* rand_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(i < n)
	{
	   if(rand_vec[i]<p)
	   {
		   in[i]=0;
	   }
	}

}

//////sigmoid
//__global__ void kernSigmoid(int n, float* in_vec, float* out_vec)
//{
//    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
//    if (i < n)
//			out_vec[i] = 1.0f/(1.0f + expf(- in_vec[i]));
//}

//__global__ void kernDsigmoid(int n, float* in_vec, float* out_vec)
//{
//    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
//
//    if (i<n)
//    {
//			const float y = in_vec[i];
//			out_vec[i] = (1.0f - y) * y;
//    }
//}

////////ReLU
__global__ void kernSigmoid(int n, float* in_vec, float* out_vec)
{
int i = (blockIdx.x * blockDim.x) + threadIdx.x;
if (i < n)
//sigmoid
//out_vec[i] = 1.0f/(1.0f + expf(- in_vec[i]));

  //ReLU
  if(in_vec[i]>0)
		out_vec[i]=in_vec[i];
	  else
		out_vec[i]=0.0f;
}

__global__ void kernDsigmoid(int n, float* in_vec, float* out_vec)
{
int i = (blockIdx.x * blockDim.x) + threadIdx.x;

if (i<n)
{
	//sigmoid
		//const float y = in_vec[i];
		//out_vec[i] = (1.0f - y) * y;
  
  //ReLU
  if(in_vec[i]>0)
		out_vec[i]=1.0f;
	  else
		out_vec[i]=0.0f;
}
}
//////////////////////////////////////////////////////////

__global__ void  kernSoftmax(int rows, int cols, float* in_vec, float* out_vec)
{
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (row < rows)
    {
			int i;
			const int index = row * cols;
			const float* invec = &in_vec[index];
		  float* outvec = &out_vec[index];
			const float* inptr;
			float* outptr;
		
			// First find the max of each vector
			float max;
			
			inptr = invec;
			max = *inptr++;
			for (i=cols-1; i!=0; i--)
			{
			    float val;
		
			    val = *inptr++;
			    if (val>max)
				max = val;
			}
			// Now put exp(in-max) in out
			inptr = invec;
			outptr = outvec;
			float sumexp = 0;
			for (i=cols; i!=0; i--)
			{
			    float f, e;
			    
			    f = *inptr++;
			    e = expf(f - max);
			    *outptr++ = e;
			    sumexp += e;
			}
			// Now scale the output
			float scale = 1.0f/sumexp;
			outptr = outvec;
			for (i=cols; i!=0; i--)
			{
			    *outptr = (*outptr) * scale;
			    outptr++;
			}
    }
}

__global__ void  kernLinearOutCopy(int rows, int cols, float* in_vec, float* out_vec)
{
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (row < rows)
    {
			//int i; //xuyong
			//const int index = row * cols;
			//const float* invec = &in_vec[index];
		  //float* outvec = &in_vec[index];
		  ////////////////////////////////////////////////////
		 int j;
	 	 for(j =0; j< cols;j++)
		 	out_vec[cols *row +j] = in_vec[cols *row +j];
		 	
    }
}

__global__ void kernMultiCopy(int mat_height, int vec_len,
		   float* vec, float* mat)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (col < vec_len)
    { 
			int j;
			float val = vec[col];
			float* top = &mat[col];
			for (j=mat_height; j!=0; j--)
			{
			    *top = val;
			    top += vec_len;
			}
    }
}

__global__ void kernSumcol(int rows, int cols, float* in, float* res)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (col < cols)
    {
			int j;
			const float* fromp = &in[col];
			float* top = &res[col];
			
			(*top) = (*fromp);
			fromp +=cols;
			for (j=rows-1; j!=0; j--)
			{
			    (*top) += (*fromp);
			    fromp+=cols;
			}
    }
}

__global__ void kernAccSumcol(int rows, int cols, float* in, float* res, float alpha, float beta)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (col < cols)
    {
			int j;
			const float* fromp = &in[col];
			float* top = &res[col];
			
			(*top) = (*top) *alpha + beta *(*fromp);
			fromp +=cols;
			for (j=rows-1; j!=0; j--)
			{
			    (*top) += beta *(*fromp);
			    fromp+=cols;
			}
    }
}

__global__ void kernAccSumrow(int rows, int cols, float* in, float* res, float alpha, float beta)
{
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (row < rows)
    {
			int j;
			const float* fromp = &in[row];
			float* top = &res[row];
			
			(*top) = (*top) *alpha + beta *(*fromp);
			fromp +=rows;
			for (j= cols -1; j!=0; j--)
			{
			    (*top) += beta *(*fromp);
			    fromp += rows;
			}
    }
}

__global__ void kernVecMul(int n, float* in_vec1, float* in_vec2, float* out_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i<n)
			out_vec[i] = in_vec1[i] * in_vec2[i];
}

//__global__ void kernSubIndex( int rows , int cols, const float *in_vec1, const int *in_index, float *res_vec)
__global__ void kernSubClean( int rows , int cols, const float *in_vec1, const float *in_clean, float *res_vec)
{
	 int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	 if(i < rows)
	 {
	 	 int j;
	 	 for(j =0; j< cols;j++)
		 	{ 	//res_vec[cols *i +j] = in_vec1[cols *i +j];
		 //int ind = in_index[i];
		 //res_vec[cols *i + ind] = in_vec1[cols *i +ind] - 1.0f;
		 res_vec[cols *i + j] = (2.0f/rows)*(in_vec1[cols *i +j]-in_clean[cols *i +j]);
		 //res_vec[cols *i + j] = 2.0f*(in_vec1[cols *i +j]-in_clean[cols *i +j]);
		 //printf("in kernSubClean, res_vec=%f ",res_vec[cols *i + j]);
		 }
	 }
}

__global__ void kernAccSum(int n, float* in, float* res, float beta)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(i <n)
	{
		res[i] = in[i] + beta *res[i];
	}
}

//__global__ void kernGetMaxIndex(int rows, int cols, float* invec, int* outvec)
//{
//	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
//	if(i < cols)
//	{
//		float *p = invec + rows * i;
//		int maxinx = 0;
//		float max = *p;
//		for(int j=1;j< rows;j++)
//		{
//			if(p[j] > max)
//			{
//				max = p[j];
//				maxinx = j;
//			}
//		}
//		outvec[i] = maxinx;
//	}
//}

__global__ void kernDivide(int n, float* in_vec, float* out_vec,float beta)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < n)
			out_vec[i] = in_vec[i]/beta;
}

//__global__ void kernUpdatedelta(int size, float* delta, float* weights, float* gradient, int n, float momentum, float lr, float weightcost)
//{
//    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
//    if (i < size)
//			delta[i] = momentum * delta[i] - lr * (gradient[i] / n + weightcost * weights[i]);
//}

__global__ void kernUpdatedelta(int size, float* delta, float* weights, float* gradient, int n, float momentum, float lr, float weightcost)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size)
			delta[i] = momentum * delta[i] - (1-momentum)*lr*(gradient[i] / n + weightcost * weights[i]);//3.16 dropoutÊ±Òª³Ë1-momentum
}
